#include "CudaChannel.h"
#include "CudaCommon.h"

__host__
CudaChannel::CudaChannel(size_t w, size_t h) :
    dData(nullptr),
    dWidth(nullptr),
    dHeight(nullptr),
    hWidth(w),
    hHeight(h)
{
    gpuErrchk( hipMalloc(&dData,   w * h * sizeof(float)) );
    gpuErrchk( hipMalloc(&dWidth,  sizeof(size_t)) );
    gpuErrchk( hipMalloc(&dHeight, sizeof(size_t)) );

    gpuErrchk( hipMemcpy(dWidth,  &hWidth,  sizeof(size_t), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dHeight, &hHeight, sizeof(size_t), hipMemcpyHostToDevice) );
}

__host__
CudaChannel::CudaChannel(const CudaChannel& channel) :
    dData(nullptr),
    dWidth(nullptr),
    dHeight(nullptr),
    hWidth(channel.hWidth),
    hHeight(channel.hHeight)
{
    const size_t bytes = hWidth * hHeight * sizeof(float);

    // Allocate memory
    gpuErrchk( hipMalloc(&dData,   bytes)          );
    gpuErrchk( hipMalloc(&dWidth,  sizeof(size_t)) );
    gpuErrchk( hipMalloc(&dHeight, sizeof(size_t)) );

    // Copy memory
    gpuErrchk( hipMemcpy(dData, channel.dData, bytes, hipMemcpyDeviceToDevice) );
    gpuErrchk( hipMemcpy(dWidth,  &hWidth,  sizeof(size_t), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dHeight, &hHeight, sizeof(size_t), hipMemcpyHostToDevice) );
}

__host__
CudaChannel::CudaChannel(CudaChannel&& channel) :
    dData(channel.dData),
    dWidth(channel.dWidth),
    dHeight(channel.dHeight),
    hWidth(channel.hWidth),
    hHeight(channel.hHeight)
{
    channel.dData   = nullptr;
    channel.dWidth  = nullptr;
    channel.dHeight = nullptr;
    channel.hWidth  = 0;
    channel.hHeight = 0;
}

__host__
CudaChannel& CudaChannel::operator=(const CudaChannel& orig)
{
    // Delete any old data
    gpuErrchk( hipFree(dData)   ); dData   = nullptr;
    gpuErrchk( hipFree(dWidth)  ); dWidth  = nullptr;
    gpuErrchk( hipFree(dHeight) ); dHeight = nullptr;

    hWidth  = orig.hWidth;
    hHeight = orig.hHeight;
    const size_t bytes = hWidth * hHeight * sizeof(float);

    // Allocate new memory
    gpuErrchk( hipMalloc(&dData,   bytes)          );
    gpuErrchk( hipMalloc(&dWidth,  sizeof(size_t)) );
    gpuErrchk( hipMalloc(&dHeight, sizeof(size_t)) );

    // Copy new data
    gpuErrchk( hipMemcpy(dData,   orig.dData, bytes, hipMemcpyDeviceToDevice)      );
    gpuErrchk( hipMemcpy(dWidth,  &hWidth,  sizeof(size_t), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dHeight, &hHeight, sizeof(size_t), hipMemcpyHostToDevice) );

    return *this;
}

__host__
CudaChannel& CudaChannel::operator=(CudaChannel&& orig)
{
    // Move original data
    dData   = orig.dData;
    dWidth  = orig.dWidth;
    dHeight = orig.dHeight;
    hWidth  = orig.hWidth;
    hHeight = orig.hHeight;

    // Set orig to a default state
    orig.dData   = nullptr;
    orig.dWidth  = nullptr;
    orig.dHeight = nullptr;
    orig.hWidth  = 0;
    orig.hHeight = 0;

    return *this;
}

__host__
CudaChannel::~CudaChannel()
{
    gpuErrchk( hipFree(dData)   );
    gpuErrchk( hipFree(dWidth)  );
    gpuErrchk( hipFree(dHeight) );
}

__device__
float& CudaChannel::operator()(const size_t x, const size_t y)
{
    return dData[y * width() + x];
}

__device__
const float& CudaChannel::operator()(const size_t x, const size_t y) const
{
    return dData[y * width() + x];
}

__host__
void CudaChannel::copyFrom(const Channel& channel)
{
    gpuErrchk( hipMemcpy(
        dData,
        channel.data.data(),
        width() * height() * sizeof(float),
        hipMemcpyHostToDevice
    ) );
}

__host__
void CudaChannel::copyTo(Channel& channel) const
{
    gpuErrchk( hipMemcpy(
        channel.data.data(),
        dData,
        width() * height() * sizeof(float),
        hipMemcpyDeviceToHost
    ) );
}

__host__ __device__
float* CudaChannel::data()
{
    return dData;
}

__host__ __device__
size_t CudaChannel::width()  const
{
    size_t result;
    #ifdef __CUDA_ARCH__
        result = *dWidth;
    #else
        result = hWidth;
    #endif
    return result;
}

__host__ __device__
size_t CudaChannel::height() const
{
    size_t result;
    #ifdef __CUDA_ARCH__
        result = *dHeight;
    #else
        result = hHeight;
    #endif
    return result;
}
